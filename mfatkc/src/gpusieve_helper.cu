#include "hip/hip_runtime.h"
/*
This file is part of mfaktc.
Copyright (C) 2009, 2010, 2011, 2012, 2014  Oliver Weihe (o.weihe@t-online.de)

mfaktc is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

mfaktc is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with mfaktc.  If not, see <http://www.gnu.org/licenses/>.
*/

__device__ static void create_k_deltas(unsigned int *bit_array, unsigned int bits_to_process, int *total_bit_count, unsigned short *k_deltas)
{
  int i, words_per_thread, sieve_word, k_bit_base;
  __shared__ volatile unsigned short bitcount[256];	// Each thread of our block puts bit-counts here

  // Get pointer to section of the bit_array this thread is processing.

  words_per_thread = bits_to_process / 8192;
  bit_array += blockIdx.x * bits_to_process / 32 + threadIdx.x * words_per_thread;

// Count number of bits set in this thread's word(s) from the bit_array

  bitcount[threadIdx.x] = 0;
  for (i = 0; i < words_per_thread; i++)
    bitcount[threadIdx.x] += __popc(bit_array[i]);

// Create total count of bits set in block up to and including this threads popc.
// Kudos to Rocke Verser for the population counting code.
// CAUTION:  Following requires 256 threads per block

  // First five tallies remain within one warp.  Should be in lock-step.
  if (threadIdx.x & 1)        // If we are running on any thread 0bxxxxxxx1, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[threadIdx.x - 1];

  if (threadIdx.x & 2)        // If we are running on any thread 0bxxxxxx1x, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 2) | 1];

  if (threadIdx.x & 4)        // If we are running on any thread 0bxxxxx1xx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 4) | 3];

  if (threadIdx.x & 8)        // If we are running on any thread 0bxxxx1xxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 8) | 7];

  if (threadIdx.x & 16)       // If we are running on any thread 0bxxx1xxxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 16) | 15];

  // Further tallies are across warps.  Must synchronize
  __syncthreads();
  if (threadIdx.x & 32)       // If we are running on any thread 0bxx1xxxxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 32) | 31];

  __syncthreads();
  if (threadIdx.x & 64)       // If we are running on any thread 0bx1xxxxxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 64) | 63];

  __syncthreads();
  if (threadIdx.x & 128)      // If we are running on any thread 0b1xxxxxxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[127];

  // At this point, bitcount[...] contains the total number of bits for the indexed
  // thread plus all lower-numbered threads.  I.e., bitcount[255] is the total count.

  __syncthreads();
  *total_bit_count = bitcount[255];

//POSSIBLE OPTIMIZATION - bitcounts and k_deltas could use the same memory space if we'd read bitcount into a register
// and sync threads before doing any writes to k_deltas.

//POSSIBLE SANITY CHECK -- is there any way to test if total_bit_count exceeds the amount of shared memory allocated?

// Loop til this thread's section of the bit array is finished.

  sieve_word = *bit_array;
  k_bit_base = threadIdx.x * words_per_thread * 32;
  for (i = *total_bit_count - bitcount[threadIdx.x]; ; i++) {
    int bit_to_test;

// Make sure we have a non-zero sieve word

    while (sieve_word == 0) {
      if (--words_per_thread == 0) break;
      sieve_word = *++bit_array;
      k_bit_base += 32;
    }

// Check if this thread has processed all its set bits

    if (sieve_word == 0) break;

// Find a bit to test in the sieve word

    bit_to_test = 31 - __clz(sieve_word);
    sieve_word &= ~(1 << bit_to_test);

// Copy the k value to the shared memory array

    k_deltas[i] = k_bit_base + bit_to_test;
  }

  __syncthreads();
// Here, all warps in our block have placed their candidates in shared memory.
// Now we can start TFing candidates.
}


__device__ static void create_fbase96(int96 *f_base, int96 k_base, unsigned int exp, unsigned int bits_to_process)
{
// Compute factor corresponding to first sieve bit in this block.

// Compute base k value
  k_base.d0 = __add_cc (k_base.d0, __umul32  (blockIdx.x * bits_to_process, NUM_CLASSES));
  k_base.d1 = __addc   (k_base.d1, __umul32hi(blockIdx.x * bits_to_process, NUM_CLASSES)); /* k values are limited to 64 bits */

// Compute k * exp
  f_base->d0 =                                      __umul32(k_base.d0, exp);
  f_base->d1 = __add_cc(__umul32hi(k_base.d0, exp), __umul32(k_base.d1, exp));
  f_base->d2 = __addc  (__umul32hi(k_base.d1, exp),                       0);

// Compute f_base = 2 * k * exp + 1
  shl_96(f_base);
  f_base->d0 = f_base->d0 + 1;
}


