#include "hip/hip_runtime.h"
/*
This file is part of mfaktc.
Copyright (C) 2009, 2010, 2011, 2012, 2014  Oliver Weihe (o.weihe@t-online.de)

mfaktc is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

mfaktc is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.
                                
You should have received a copy of the GNU General Public License
along with mfaktc.  If not, see <http://www.gnu.org/licenses/>.
*/


#ifndef DEBUG_GPU_MATH
  #ifdef INV_160_96
__device__ static void inv_160_96(int96 *res, int96 n, float nf)
  #else
__device__ static void div_192_96(int96 *res, int192 q, int96 n, float nf)
  #endif
#else
  #ifdef INV_160_96
__device__ static void inv_160_96(int96 *res, int96 n, float nf, unsigned int *modbasecase_debug)
  #else
__device__ static void div_192_96(int96 *res, int192 q, int96 n, float nf, unsigned int *modbasecase_debug)
  #endif
#endif
/* res = q / n (integer division) */
/* the code of inv_160_96() is an EXACT COPY of div_192_96(), the only
difference is that the 160bit version assumes q = 2^160. This is controlled
by defining INV_160_96 here.

Input assumptions:
- div_192_96()
  - q = 2^(2*ceil(log2(n))) (~ twize the size of n, if 2^74 < n < 2^75 than q = 2^150)
  - 2^64 < n < 2^96
  -> 2^64 < res < 2^96
  
- div_160_96()
  - q = 2^160 (implicit, not a real parameter)
  - 2^64 < n < 2^80
  -> 2^80 < res < 2^96
*/
{
  float qf;
  unsigned int qi;
  int192 nn;
#ifdef DEBUG_GPU_MATH
  int96 tmp96;
#endif
  
#ifdef INV_160_96
  int192 q;
// set q to (nearly 2^160)
  q.d5 = 0;
  q.d4 = 0xFFFFFFFF;
  q.d3 = 0xFFFFFFFF;
  q.d2 = 0xFFFFFFFF;
  q.d1 = 0xFFFFFFFF;
  q.d0 = 0xFFFFFFFF;
#endif

/********** Step 1, Offset 2^75 (2*32 + 11) **********/
#ifndef INV_160_96
  qf= __uint2float_rn(q.d5);
  qf= qf * 4294967296.0f + __uint2float_rn(q.d4);
  qf*= 2097152.0f;
#else
  qf = 9007199254740992.0f; // 2^(32 + (32 - 11))
#endif  

  qi=__float2uint_rz(qf*nf);

// EDIT OW 2014-10-20: qi can't be > 2^21, otherwise result would be > 2^96... so check for 2^21 insteat of 2^22 here
// this allows/avoid long shift after multiplication, we can do a simple shift before multiplication.  
  MODBASECASE_QI_ERROR(1<<21, 1, qi, 0);

#if __CUDA_ARCH__ >= KEPLER
  qi = __umul32(qi, 2048);
#else
  qi <<= 11;
#endif
  
  res->d2 = qi;

// nn = n * qi
  nn.d2 =                                 __umul32(n.d0, qi);
#if (__CUDA_ARCH__ >= KEPLER) && (CUDART_VERSION >= 4010) /* multiply-add with carry is not available on CC 1.x devices and before CUDA 4.1 */
  nn.d3 = __umad32hi_cc       (n.d0, qi,  __umul32(n.d1, qi));
  #ifndef INV_160_96
  nn.d4 = __umad32hic_cc      (n.d1, qi,  __umul32(n.d2, qi));
  nn.d5 = __umad32hic         (n.d2, qi,                  0);
  #else
  nn.d4 = __umad32hic         (n.d1, qi,  __umul32(n.d2, qi));
  #endif
#else
  nn.d3 = __add_cc (__umul32hi(n.d0, qi), __umul32(n.d1, qi));
  #ifndef INV_160_96
  nn.d4 = __addc_cc(__umul32hi(n.d1, qi), __umul32(n.d2, qi));
  nn.d5 = __addc   (__umul32hi(n.d2, qi),                  0);
  #else
  nn.d4 = __addc   (__umul32hi(n.d1, qi), __umul32(n.d2, qi));
  #endif
#endif

//  q = q - nn
  q.d2 = __sub_cc (q.d2, nn.d2);
  q.d3 = __subc_cc(q.d3, nn.d3);
#ifndef INV_160_96
  q.d4 = __subc_cc(q.d4, nn.d4);
  q.d5 = __subc   (q.d5, nn.d5);
#else
  q.d4 = __subc   (q.d4, nn.d4);
#endif

/********** Step 2, Offset 2^55 (1*32 + 23) **********/
#ifndef INV_160_96
  qf= __uint2float_rn(q.d5);
  qf= qf * 4294967296.0f + __uint2float_rn(q.d4);
#else
  qf= __uint2float_rn(q.d4);
#endif
  qf= qf * 4294967296.0f + __uint2float_rn(q.d3);
  qf*= 512.0f;

  qi=__float2uint_rz(qf*nf);

  MODBASECASE_QI_ERROR(1<<22, 2, qi, 1);

#if __CUDA_ARCH__ >= KEPLER
  res->d1 =  __umul32(qi, 8388608);
#else
  res->d1 =  qi << 23;
#endif
  res->d2 += qi >>  9;

// nn = n * qi
  nn.d1 =                                 __umul32(n.d0, qi);
  nn.d2 = __add_cc (__umul32hi(n.d0, qi), __umul32(n.d1, qi));
  nn.d3 = __addc_cc(__umul32hi(n.d1, qi), __umul32(n.d2, qi));
  nn.d4 = __addc   (__umul32hi(n.d2, qi),                  0);

// shiftleft nn 23 bits
#ifdef DEBUG_GPU_MATH
  nn.d5 =                  nn.d4 >> 9;
#endif  
#if __CUDA_ARCH__ >= KEPLER
  nn.d4 = __umad32(nn.d4, 8388608, (nn.d3 >> 9));
  nn.d3 = __umad32(nn.d3, 8388608, (nn.d2 >> 9));
  nn.d2 = __umad32(nn.d2, 8388608, (nn.d1 >> 9));
  nn.d1 = __umul32(nn.d1, 8388608);
#else
  nn.d4 = (nn.d4 << 23) + (nn.d3 >> 9);
  nn.d3 = (nn.d3 << 23) + (nn.d2 >> 9);
  nn.d2 = (nn.d2 << 23) + (nn.d1 >> 9);
  nn.d1 =  nn.d1 << 23;
#endif

// q = q - nn
  q.d1 = __sub_cc (q.d1, nn.d1);
  q.d2 = __subc_cc(q.d2, nn.d2);
  q.d3 = __subc_cc(q.d3, nn.d3);
#ifndef DEBUG_GPU_MATH  
  q.d4 = __subc   (q.d4, nn.d4);
#else
  q.d4 = __subc_cc(q.d4, nn.d4);
  q.d5 = __subc   (q.d5, nn.d5);
#endif

/********** Step 3, Offset 2^35 (1*32 + 3) **********/
  MODBASECASE_NONZERO_ERROR(q.d5, 3, 5, 2);

  qf= __uint2float_rn(q.d4);
  qf= qf * 4294967296.0f + __uint2float_rn(q.d3);
  qf*= 536870912.0f; // add (q.d1 >> 3) ???
//  qf*= 4294967296.0f; /* this includes the shiftleft of qi by 3 bits! */

  qi=__float2uint_rz(qf*nf);

  MODBASECASE_QI_ERROR(1<<22, 3, qi, 3);

#if __CUDA_ARCH__ >= KEPLER
  res->d1 = __add_cc(res->d1, __umul32(qi, 8) );
#else
  res->d1 = __add_cc(res->d1, qi << 3 );
#endif
  res->d2 = __addc  (res->d2, qi >> 29);

// shiftleft qi 3 bits to avoid "long shiftleft" after multiplication
#if __CUDA_ARCH__ >= KEPLER
  qi *= 8;
#else
  qi <<= 3;
#endif

// nn = n * qi
  nn.d1 =                                 __umul32(n.d0, qi);
  nn.d2 = __add_cc (__umul32hi(n.d0, qi), __umul32(n.d1, qi));
  nn.d3 = __addc_cc(__umul32hi(n.d1, qi), __umul32(n.d2, qi));
  nn.d4 = __addc   (__umul32hi(n.d2, qi),                  0);

//  q = q - nn
  q.d1 = __sub_cc (q.d1, nn.d1);
  q.d2 = __subc_cc(q.d2, nn.d2);
  q.d3 = __subc_cc(q.d3, nn.d3);
  q.d4 = __subc   (q.d4, nn.d4);

/********** Step 4, Offset 2^15 (0*32 + 15) **********/
  MODBASECASE_NONZERO_ERROR(q.d5, 4, 5, 4);

  qf= __uint2float_rn(q.d4);
  qf= qf * 4294967296.0f + __uint2float_rn(q.d3);
  qf= qf * 4294967296.0f + __uint2float_rn(q.d2);
  qf*= 131072.0f;
  
  qi=__float2uint_rz(qf*nf);

  MODBASECASE_QI_ERROR(1<<22, 4, qi, 5);

#if __CUDA_ARCH__ >= KEPLER
  res->d0 = __umul32(qi, 32768);
#else
  res->d0 = qi << 15;
#endif
  res->d1 = __add_cc(res->d1, qi >> 17);
  res->d2 = __addc  (res->d2, 0);
  
// nn = n * qi
  nn.d0 =                                 __umul32(n.d0, qi);
  nn.d1 = __add_cc (__umul32hi(n.d0, qi), __umul32(n.d1, qi));
  nn.d2 = __addc_cc(__umul32hi(n.d1, qi), __umul32(n.d2, qi));
  nn.d3 = __addc   (__umul32hi(n.d2, qi),                  0);

// shiftleft nn 15 bits
#ifdef DEBUG_GPU_MATH
  nn.d4 =                  nn.d3 >> 17;
#endif
#if __CUDA_ARCH__ >= KEPLER
  nn.d3 = __umad32(nn.d3, 32768, (nn.d2 >> 17));
  nn.d2 = __umad32(nn.d2, 32768, (nn.d1 >> 17));
  nn.d1 = __umad32(nn.d1, 32768, (nn.d0 >> 17));
  nn.d0 = __umul32(nn.d0, 32768);
#else
  nn.d3 = (nn.d3 << 15) + (nn.d2 >> 17);
  nn.d2 = (nn.d2 << 15) + (nn.d1 >> 17);
  nn.d1 = (nn.d1 << 15) + (nn.d0 >> 17);
  nn.d0 =  nn.d0 << 15;
#endif

//  q = q - nn
  q.d0 = __sub_cc (q.d0, nn.d0);
  q.d1 = __subc_cc(q.d1, nn.d1);
  q.d2 = __subc_cc(q.d2, nn.d2);
#ifndef DEBUG_GPU_MATH
  q.d3 = __subc   (q.d3, nn.d3);
#else
  q.d3 = __subc_cc(q.d3, nn.d3);
  q.d4 = __subc   (q.d4, nn.d4);
#endif

/********** Step 5, Offset 2^0 (0*32 + 0) **********/
  MODBASECASE_NONZERO_ERROR(q.d5, 5, 5, 6);
  MODBASECASE_NONZERO_ERROR(q.d4, 5, 4, 7);

  qf= __uint2float_rn(q.d3);
  qf= qf * 4294967296.0f + __uint2float_rn(q.d2);
  qf= qf * 4294967296.0f + __uint2float_rn(q.d1);
  
  qi=__float2uint_rz(qf*nf);

  MODBASECASE_QI_ERROR(1<<20, 5, qi, 8);

  res->d0 = __add_cc (res->d0, qi);
  res->d1 = __addc_cc(res->d1,  0);
  res->d2 = __addc   (res->d2,  0);
  
#ifdef DEBUG_GPU_MATH
/* compute to the end only in DEBUG_GPU_MATH mode */

// nn = n * qi
  nn.d0 =                                 __umul32(n.d0, qi);
  nn.d1 = __add_cc (__umul32hi(n.d0, qi), __umul32(n.d1, qi));
  nn.d2 = __addc_cc(__umul32hi(n.d1, qi), __umul32(n.d2, qi));
  nn.d3 = __addc   (__umul32hi(n.d2, qi),                  0);

//  q = q - nn
  q.d0 = __sub_cc (q.d0, nn.d0);
  q.d1 = __subc_cc(q.d1, nn.d1);
  q.d2 = __subc_cc(q.d2, nn.d2);
  q.d3 = __subc   (q.d3, nn.d3);

  tmp96.d0=q.d0;
  tmp96.d1=q.d1;
  tmp96.d2=q.d2;

  if(n.d2) /* only care about errors is n >= 2^64 (see function description) */
  {  
    MODBASECASE_NONZERO_ERROR(q.d5, 6, 5, 9);
    MODBASECASE_NONZERO_ERROR(q.d4, 6, 4, 10);
    MODBASECASE_NONZERO_ERROR(q.d3, 6, 3, 11);
  }

/*
qi is allways a little bit too small, this is OK for all steps except the last
one. Sometimes the result is a little bit bigger than n
*/
  if(cmp_ge_96(tmp96,n))
  {
/*    res->d0 = __add_cc (res->d0,  1);
    res->d1 = __addc_cc(res->d1,  0);
    res->d2 = __addc   (res->d2,  0);*/
    tmp96.d0 = __sub_cc (tmp96.d0, n.d0);
    tmp96.d1 = __subc_cc(tmp96.d1, n.d1);
    tmp96.d2 = __subc   (tmp96.d2, n.d2);
  }
  if(cmp_ge_96(tmp96,n))
  {
  #if defined USE_DEVICE_PRINTF && __CUDA_ARCH__ >= FERMI
    printf("EEEEEK, final value of tmp96 is too big\n");
    printf("  tmp96 = 0x %08x %08x %08x\n", tmp96.d2, tmp96.d1, tmp96.d0);
    printf("  n =     0x %08x %08x %08x\n", n.d2, n.d1, n.d0);
  #endif
  }
#endif
}
