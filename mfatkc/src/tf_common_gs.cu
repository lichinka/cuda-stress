/*
This file is part of mfaktc.
Copyright (C) 2009, 2010, 2011, 2012, 2014, 2015  Oliver Weihe (o.weihe@t-online.de)

mfaktc is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

mfaktc is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with mfaktc.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "gpusieve.h"

#undef RAW_GPU_BENCH // FIXME


#ifdef TF_96BIT
  #ifdef SHORTCUT_75BIT
extern "C" __host__ int tf_class_75_gs(unsigned long long int k_min, unsigned long long int k_max, mystuff_t *mystuff)
#define MFAKTC_FUNC mfaktc_75_gs
  #else
extern "C" __host__ int tf_class_95_gs(unsigned long long int k_min, unsigned long long int k_max, mystuff_t *mystuff)
#define MFAKTC_FUNC mfaktc_95_gs
  #endif
#endif
#ifdef TF_BARRETT
  #ifdef TF_BARRETT_76BIT_GS
extern "C" __host__ int tf_class_barrett76_gs(unsigned long long int k_min, unsigned long long int k_max, mystuff_t *mystuff)
#define MFAKTC_FUNC mfaktc_barrett76_gs
  #elif defined TF_BARRETT_77BIT_GS
extern "C" __host__ int tf_class_barrett77_gs(unsigned long long int k_min, unsigned long long int k_max, mystuff_t *mystuff)
#define MFAKTC_FUNC mfaktc_barrett77_gs
  #elif defined TF_BARRETT_79BIT_GS
extern "C" __host__ int tf_class_barrett79_gs(unsigned long long int k_min, unsigned long long int k_max, mystuff_t *mystuff)
#define MFAKTC_FUNC mfaktc_barrett79_gs
  #elif defined TF_BARRETT_87BIT_GS
extern "C" __host__ int tf_class_barrett87_gs(unsigned long long int k_min, unsigned long long int k_max, mystuff_t *mystuff)
#define MFAKTC_FUNC mfaktc_barrett87_gs
  #elif defined TF_BARRETT_88BIT_GS
extern "C" __host__ int tf_class_barrett88_gs(unsigned long long int k_min, unsigned long long int k_max, mystuff_t *mystuff)
#define MFAKTC_FUNC mfaktc_barrett88_gs
  #else
extern "C" __host__ int tf_class_barrett92_gs(unsigned long long int k_min, unsigned long long int k_max, mystuff_t *mystuff)
#define MFAKTC_FUNC mfaktc_barrett92_gs
  #endif
#endif
{
  int i;
  timeval timer;
  int96 factor,k_base;
  int192 b_preinit;
  int shiftcount, ln2b, count = 0;
  int numblocks;
  unsigned long long k_remaining;
  char string[50];
  int shared_mem_required;
  int factorsfound = 0;

  // If we've never initialized the GPU sieving code, do so now.
//  gpusieve_init (mystuff); // moved to main() function!

  // If we haven't initialized the GPU sieving code for this Mersenne exponent, do so now.
  gpusieve_init_exponent (mystuff);

  // Init the timer
  timer_init(&timer);

  // Pre-calculate some values

  shiftcount=0;
  while((1ULL<<shiftcount) < (unsigned long long int)mystuff->exponent)shiftcount++;
//  printf("\n\nshiftcount = %d\n",shiftcount);
  shiftcount-=1;ln2b=1;
  while(ln2b<20 || ln2b<mystuff->bit_min)	// how much preprocessing is possible
  {
    shiftcount--;
    ln2b<<=1;
    if(mystuff->exponent&(1<<(shiftcount)))ln2b++;
  }
//  printf("shiftcount = %d\n",shiftcount);
//  printf("ln2b = %d\n",ln2b);
  b_preinit.d5=0;b_preinit.d4=0;b_preinit.d3=0;b_preinit.d2=0;b_preinit.d1=0;b_preinit.d0=0;
  if     (ln2b<32 )b_preinit.d0=1<< ln2b;
  else if(ln2b<64 )b_preinit.d1=1<<(ln2b-32);
  else if(ln2b<96 )b_preinit.d2=1<<(ln2b-64);
  else if(ln2b<128)b_preinit.d3=1<<(ln2b-96);
  else if(ln2b<160)b_preinit.d4=1<<(ln2b-128);
  else             b_preinit.d5=1<<(ln2b-160);	// b_preinit = 2^ln2b

/* set result array to 0 */  
  hipMemset(mystuff->d_RES, 0, 1*sizeof(int)); //first int of result array contains the number of factors found

#ifdef DEBUG_GPU_MATH  
  hipMemset(mystuff->d_modbasecase_debug, 0, 32*sizeof(int));
#endif

  // Calculate the initial bit-to-clear values for this class
  gpusieve_init_class (mystuff, k_min);

  // Generously estimate the shared memory requirements for the TF kernel
#ifdef RAW_GPU_BENCH
  shared_mem_required = 100;						// no sieving = 100%
#else
  if (mystuff->gpu_sieve_primes < 54) shared_mem_required = 100;	// no sieving = 100%
  else if (mystuff->gpu_sieve_primes < 310) shared_mem_required = 50;	// 54 primes expect 48.30%
  else if (mystuff->gpu_sieve_primes < 1846) shared_mem_required = 38;	// 310 primes expect 35.50%
  else if (mystuff->gpu_sieve_primes < 21814) shared_mem_required = 30;	// 1846 primes expect 28.10%
  else if (mystuff->gpu_sieve_primes < 67894) shared_mem_required = 24;	// 21814 primes expect 21.93%
  else shared_mem_required = 22;					// 67894 primes expect 19.94%
#endif
  shared_mem_required = mystuff->gpu_sieve_processing_size * sizeof (int) * shared_mem_required / 100;
  
  // FIXME: can't use all the shared memory for GPU sieve, lets keep 1kiB spare...
  if(mystuff->verbosity >= 3)printf("shared_mem_required = %d bytes\n", shared_mem_required + 1024);

  if((shared_mem_required + 1024) > mystuff->max_shared_memory)
  {
    printf("ERROR: Not enough shared memory available!\n");
    printf("       Need %d bytes\n", shared_mem_required + 1024);
    printf("       You can lower GPUSieveProcessSize or increase GPUSievePrimes to lower\n");
    printf("       the amount of shared memory needed\n");
    exit(1);
  }
     

  // Loop until all the k's are processed
  for(;;)
  {

    // Calculate the number of k's remaining.  Round this up so that we sieve an array that is
    // a multiple of the bits processed by each TF kernel (my_stuff->gpu_sieve_processing_size).

    k_remaining = ((k_max - k_min + 1) + NUM_CLASSES - 1) / NUM_CLASSES;
    if (k_remaining < (unsigned long long) mystuff->gpu_sieve_size) {
      numblocks = ((int) k_remaining + mystuff->gpu_sieve_processing_size - 1) / mystuff->gpu_sieve_processing_size;
      k_remaining = numblocks * mystuff->gpu_sieve_processing_size;
    } else
      numblocks = mystuff->gpu_sieve_size / mystuff->gpu_sieve_processing_size;

    // Do some sieving on the GPU.

    gpusieve (mystuff, k_remaining);

    // Set the k value corresponding to the first bit in the bit array

    k_base.d0 = (int) (k_min & 0xFFFFFFFF);
    k_base.d1 = (int) (k_min >> 32);
    k_base.d2 = 0;

    // Now let the GPU trial factor the candidates that survived the sieving

    MFAKTC_FUNC<<<numblocks, THREADS_PER_BLOCK, shared_mem_required>>>(mystuff->exponent, k_base, mystuff->d_bitarray, mystuff->gpu_sieve_processing_size, shiftcount, b_preinit, mystuff->d_RES
#if defined (TF_BARRETT) && (defined(TF_BARRETT_87BIT_GS) || defined(TF_BARRETT_88BIT_GS) || defined(TF_BARRETT_92BIT_GS) || defined(DEBUG_GPU_MATH))
                                                                       , mystuff->bit_min-63
#endif
#ifdef DEBUG_GPU_MATH
                                                                       , mystuff->d_modbasecase_debug
#endif
                                                                       );

    // Sync before doing more GPU sieving
    hipDeviceSynchronize();

    // Count the number of blocks processed
    count += numblocks;

    // Move to next batch of k's
    k_min += (unsigned long long) mystuff->gpu_sieve_size * NUM_CLASSES;
    if (k_min > k_max) break;

    //BUG - we should call a different routine to advance the bit-to-clear values by gpusieve_size bits
    // This will be cheaper than recomputing the bit-to-clears from scratch
    // HOWEVER, the self-test code will ot check this new code unless we make the gpusieve_size much smaller
    gpusieve_init_class (mystuff, k_min);
  }

/* download results from GPU */
  hipMemcpy(mystuff->h_RES, mystuff->d_RES, 32*sizeof(int), hipMemcpyDeviceToHost);

#ifdef DEBUG_GPU_MATH
  hipMemcpy(mystuff->h_modbasecase_debug, mystuff->d_modbasecase_debug, 32*sizeof(int), hipMemcpyDeviceToHost);
  for(i=0;i<32;i++)if(mystuff->h_modbasecase_debug[i] != 0)printf("h_modbasecase_debug[%2d] = %u\n", i, mystuff->h_modbasecase_debug[i]);
#endif  

  // Set grid count to the number of blocks processed.  The print code will convert this to a
  // count of candidates processed (by multiplying by 8192 * THREADS_PER_BLOCK.
  // This count isn't an exact match to CPU sieving case as that counts candidates after sieving
  // and we are counting candidates before sieving.  We'd have to modify the TF kernels to count
  // the candidates processed to be completely compatible.
  mystuff->stats.grid_count = count;

  // Keep track of time spent TFing this class
  /* prevent division by zero if timer resolution is too low */
  mystuff->stats.class_time = timer_diff(&timer)/1000;
  if(mystuff->stats.class_time == 0)mystuff->stats.class_time = 1;

  // GPU sieving does not wait on the CPU (also used by print_status_line to indicate this is a GPU sieving kernel)
  mystuff->stats.cpu_wait = -2.0f;

  // Print out a useful status line
  print_status_line(mystuff);

  // Print out any found factors
  factorsfound=mystuff->h_RES[0];
  for(i=0; (i<factorsfound) && (i<10); i++)
  {
    factor.d2=mystuff->h_RES[i*3 + 1];
    factor.d1=mystuff->h_RES[i*3 + 2];
    factor.d0=mystuff->h_RES[i*3 + 3];
    print_dez96(factor,string);
    print_factor(mystuff, i, string);
  }
  if(factorsfound>=10)
  {
    print_factor(mystuff, factorsfound, NULL);
  }

  return factorsfound;
}

#undef MFAKTC_FUNC
