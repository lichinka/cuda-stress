
#include <hip/hip_runtime.h>
// Actually, there are no rounding errors due to results being accumulated in an arbitrary order..
// Therefore EPSILON = 0.0f is OK
#define EPSILON 0.001f
#define EPSILOND 0.0000001

extern "C" __global__ void compare(float *C, int *faultyElems, size_t iters) {
	size_t iterStep = blockDim.x*blockDim.y*gridDim.x*gridDim.y;
	size_t myIndex = (blockIdx.y*blockDim.y + threadIdx.y)* // Y
		gridDim.x*blockDim.x + // W
		blockIdx.x*blockDim.x + threadIdx.x; // X

	int myFaulty = 0;
	for (size_t i = 1; i < iters; ++i)
		if (fabsf(C[myIndex] - C[myIndex + i*iterStep]) > EPSILON)
			myFaulty++;

	atomicAdd(faultyElems, myFaulty);
}

extern "C" __global__ void compareD(double *C, int *faultyElems, size_t iters) {
	size_t iterStep = blockDim.x*blockDim.y*gridDim.x*gridDim.y;
	size_t myIndex = (blockIdx.y*blockDim.y + threadIdx.y)* // Y
		gridDim.x*blockDim.x + // W
		blockIdx.x*blockDim.x + threadIdx.x; // X

	int myFaulty = 0;
	for (size_t i = 1; i < iters; ++i)
		if (fabs(C[myIndex] - C[myIndex + i*iterStep]) > EPSILOND)
			myFaulty++;

	atomicAdd(faultyElems, myFaulty);
}
